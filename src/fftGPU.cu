/* Raul P. Pelaez 2020. Fast Fourier Transform cuFFT */
#include"fftGPU.h"
#include<hipfft/hipfft.h>
#include<thrust/device_vector.h>

namespace gpu_mode{
  
  template<class T> struct CUFFT_Real2Complex;
  template<> struct CUFFT_Real2Complex<double>{static constexpr hipfftType value=HIPFFT_D2Z;};
  template<> struct CUFFT_Real2Complex<float>{static constexpr hipfftType value=HIPFFT_R2C;};

  template<class floatType>
  void cufftExecReal2Complex(hipfftHandle &plan, cufftReal_t<floatType>* d_in, cufftComplex_t<floatType> *d_out);
  
  template<>
  void cufftExecReal2Complex<float>(hipfftHandle &plan, cufftReal_t<float>* d_in, cufftComplex_t<float> *d_out){
    hipfftExecR2C(plan, d_in, d_out);
  }

  template<>
  void cufftExecReal2Complex<double>(hipfftHandle &plan, cufftReal_t<double>* d_in, cufftComplex_t<double> *d_out){
    hipfftExecD2Z(plan, d_in, d_out);
  }
}

template<class floatType>
std::vector<gpu_mode::cufftComplex_t<floatType>> computeFFTWithCUFFT(const std::vector<floatType> &signal, Config config){
  using cufftComplex_t = gpu_mode::cufftComplex_t<floatType>;
  using cufftReal_t = gpu_mode::cufftReal_t<floatType>;
  thrust::device_vector<cufftComplex_t> data(config.numberElements/2+1);
  cufftComplex_t* d_m = thrust::raw_pointer_cast(data.data());
  thrust::device_ptr<floatType> d_data((floatType*) d_m);
  thrust::copy(signal.begin(), signal.end(), d_data);
  hipfftHandle plan;
  hipfftPlan1d(&plan, config.numberElements, gpu_mode::CUFFT_Real2Complex<floatType>::value, 1);
  gpu_mode::cufftExecReal2Complex<floatType>(plan, (cufftReal_t*) d_m, d_m);
  std::vector<cufftComplex_t> fft(config.numberElements/2+1);
  thrust::copy(data.begin(), data.end(), fft.begin());
  return std::move(fft);
}

template std::vector<gpu_mode::cufftComplex_t<float>> computeFFTWithCUFFT(const std::vector<float> &signal, Config config);
template std::vector<gpu_mode::cufftComplex_t<double>> computeFFTWithCUFFT(const std::vector<double> &signal, Config config);
