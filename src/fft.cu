#include "hip/hip_runtime.h"
/* Raul P. Pelaez 2017. Fast Fourier Transform

   Takes a single column signal and outputs its FFT (using cuda) in frequency domain.
   The output format is:
      k Amplitude(k) phase(k)
      
Usage:

fft -N [N] -Fs [Fs] -prec [double/float] < signal

N: Number of points in the signal
Fs: Sampling frequency
prec: Precision mode, float by default

Example:

seq 0 0.2 10000 |  awk '{print sin($1)}' | fft -N 10000 -Fs 5.0 > kk

w=$(grep $( datamash -W max 2 <kk) kk | awk '{print 2*3.1415*$1}')

w will be 1

 */
#include<iostream>

#include<vector>

#include"fftGPU.cuh"

//#include"fftCPU.h"

#include<cmath>
#include<sstream>

//Ask for one or many arguments of any type after a flag from argv
template<class T>
bool parseArgument(int argc, char *argv[],
		   const char *flag,  //What to look for
		   T* result, int numberArguments=1){ //output and number of them
  for(int i=1; i<argc; i++){ //Look for the flag

    if(strcmp(flag, argv[i]) == 0){ //If found it
      std::string line;  //convert char * to string for as many values as requested
      for(int j=0; j<numberArguments; j++)
	line += argv[i+j+1];

      std::istringstream ss(line);
      //Store them in result
      for(int j=0; j<numberArguments; j++){
	ss>>result[j];
      }
      return true;
    }
  }
  return false;
}


void print_help();
int main(int argc, char *argv[]){
  if(argc<3){
    print_help();
    exit(1);
  }
  
  int numberElements;
  if(!parseArgument(argc, argv, "-N", &numberElements)){
    print_help();
    exit(1);
  }
    
  double Fs;
  if(!parseArgument(argc, argv, "-Fs", &Fs)){
    print_help();
    exit(1);
  }

  std::string prec = "float";
  parseArgument(argc, argv, "-prec", &prec);

  std::string mode = "GPU";
  parseArgument(argc, argv, "-mode", &mode);

  if(prec.compare("float")==0){
    if(mode.compare("GPU")==0){      
      FFTCudaMode<float>(numberElements, Fs);
    }
    else if(mode.compare("CPU")==0){
    }
    else{
      std::cerr<<"Unrecognized mode"<<std::endl;
      print_help();
      exit(1);
    }
  }
  else if(prec.compare("double")==0){
    if(mode.compare("GPU")==0){
      FFTCudaMode<double>(numberElements, Fs);
    }
    else if(mode.compare("CPU")==0){
    }
    else{
      std::cerr<<"Unrecognized mode"<<std::endl;
      print_help();
      exit(1);
    }	
  }
  else{
    std::cerr<<"Unrecognized prec"<<std::endl;
    print_help();
    exit(1);    
  }
  
   
 return 0;
}


void print_help(){
std::cerr<<"ERROR!!: Input missing"<<std::endl;
    std::cerr<<"Takes a single column signal and outputs its FFT (using cuda) in frequency domain."<<std::endl;
    std::cerr<<"The output format is:"<<std::endl;
    std::cerr<<"\tk Amplitude(k) phase(k)"<<std::endl;

    std::cerr<<""<<std::endl;
    std::cerr<<"Usage:"<<std::endl;
    std::cerr<<""<<std::endl;
    std::cerr<<"fft -N [N] -Fs [Fs] -prec [float/double] < signal"<<std::endl;
    std::cerr<<""<<std::endl;
    std::cerr<<"N: Number of points in the signal"<<std::endl;
    std::cerr<<"Fs: Sampling frequency"<<std::endl;
    std::cerr<<"prec: Precision mode, double by default"<<std::endl;
    std::cerr<<""<<std::endl;
    std::cerr<<"Example:"<<std::endl;
    std::cerr<<""<<std::endl;
    std::cerr<<"seq 0 0.2 10000 |  awk '{print sin($1)+rand()-0.5}' | fft -N 10000 -Fs 5.0 > kk"<<std::endl;
    std::cerr<<""<<std::endl;
    std::cerr<<"w=$(grep $( datamash -W max 2 <kk) kk | awk '{print 2*3.1415*$1}')"<<std::endl;
    std::cerr<<std::endl;
    std::cerr<<"w will be close to 1"<<std::endl;    
}
